#include<stdio.h>
#include<hip/hip_runtime.h>

//kernel definition
__global__ void increment_gpu(int* a, int* b, int* c, int N)
{
	int i = blockIdx.x;
	if (i < N)
		c[i] = a[i] + b[i];
}

int main()
{
	int n;
	printf("Enter n\n");
	scanf("%d", &n);
	int N = n;
	int h1_a[100], h2_a[100];
	//int h2_a[5] = { 10,20,30,40,50 };
	int h3_a[100] = { 0 };

	printf("Enter the 1st array\n");
	for (int i = 0; i < n; i++)
		scanf("%d", &h1_a[i]);
	printf("Enter the 2nd array\n");
	for(int i = 0; i < n; i++)
		scanf("%d", &h2_a[i]);
	int* d1_a;
	int* d2_a;
	int* d3_a;
	hipMalloc((void**)&d1_a, N * sizeof(int));
	hipMalloc((void**)&d2_a, N * sizeof(int));
	hipMalloc((void**)&d3_a, N * sizeof(int));
	hipMemcpy(d1_a, h1_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d2_a, h2_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d3_a, h3_a, N * sizeof(int), hipMemcpyHostToDevice);

	dim3 grid_size(N);
	dim3 block_size(1);

	increment_gpu <<<grid_size, block_size>>>(d1_a, d2_a, d3_a, N);

	hipMemcpy(h3_a, d3_a, N * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d1_a);
	hipFree(d2_a);
	hipFree(d3_a);
	for (int i = 0; i < N; i++)
	{
		printf("%d ", h3_a[i]);
	}
	return 0;
}